#include "hip/hip_runtime.h"
#include "CollisionCalculator.cuh"
#include "hip/hip_runtime.h"
#include ""
#include "Global_Flags.h"
static int ArraySize = COLLISION_INIT_SIZE;
struct GameObjectCUDA {
	float2 points[4];
};





void CalculateCollsion(std::vector<GameObject*>& UpdateQueue)
{
	while (ArraySize > UpdateQueue.size()) {
		ArraySize += COLLISION_EXPAND_SIZE;
	}

}