#include "hip/hip_runtime.h"
#include "CollisionCalculator.cuh"
#include "hip/hip_runtime.h"
#include ""
#include "Global_Flags.h"
#include "GameMath.h"
#include <math.h>
#include "GameLogging.h"
static int AllocSize = COLLISION_INIT_SIZE;

struct GameObjectCUDA {
	int id;
	/*
	* points MUST be fed as
	* (technically order doesnt matter but 0 and 3 must be opposite)
	0-------1
	|       |
	|       |
	2-------3
	*/
	float2 points[4];
	float2 centre;
	int colliders[MAX_COLLISIONS];
	unsigned int currentIdx = 0;
};
static GameObjectCUDA* objsGPU;
static bool allocated = false;
__device__ inline float dotProduct(float2 a, float2 b) {
	return (a.x * b.x) + (a.y * b.y);
}
__device__ inline float2 CalcNormalAxis(float2 p1, float2 p2) {
	//return -dy, dx
	float dy = p1.y - p2.y;
	float dx = p1.x - p2.x;
	float length = sqrtf(dx * dx + dy * dy);
	float2 axis = make_float2(dy / (length * -1), dx / length);
	return axis;
}
__device__ inline float2 MinMaxProjectionOnAxis(float2 points[4],float2 axis) {
	//https://textbooks.cs.ksu.edu/cis580/04-collisions/04-separating-axis-theorem/index.html
	//accessed 19/04/24
	//project each point
	//return min,max
	float projection = dotProduct(points[0],axis);
	float min = projection;
	float max = projection;
	for (int i = 1; i < 4; i++) {
		projection = dotProduct(points[i], axis);
		max = max > projection ? max : projection;
		min = min < projection ? min : projection;
	}
	return make_float2(min, max);
}
__device__ inline bool SATCheck(GameObjectCUDA* object, GameObjectCUDA* other) {
	//TRUE IS A COLLISION
	//Last step -- implement actual alg
	//we are assuming boxes so we only need up normal and right normal for both
	float2 normals[4];
	normals[0] = CalcNormalAxis(object->points[0], object->points[1]);
	normals[1] = CalcNormalAxis(object->points[1], object->points[3]);
	normals[2] = CalcNormalAxis(other->points[0], other->points[1]);
	normals[3] = CalcNormalAxis(other->points[1], other->points[3]);
	for (int i = 0; i < 4; i++) {
		float2 objectProjection = MinMaxProjectionOnAxis(object->points, normals[i]);
		float2 otherProjection = MinMaxProjectionOnAxis(other->points, normals[i]);
		if (objectProjection.x > otherProjection.y || otherProjection.x > objectProjection.y) {
			return false;
		}
	}
	return true;
}
__device__ inline float SqrDistance(float2 p1, float2 p2) {
	float2 offset = make_float2(p1.x - p2.x, p1.y - p2.y);
	return (offset.x * offset.x) + (offset.y * offset.y);
}
__device__ inline bool SphereCheck(GameObjectCUDA* object, GameObjectCUDA* other) {
	float SQRradius = fmaxf(SqrDistance(object->centre,object->points[0]), SqrDistance(other->centre,other->points[0]));

	return SqrDistance(object->centre, other->centre) < SQRradius;
}
__global__ void GPUCollisionCalc(GameObjectCUDA* objs, int size) {
	int i = blockIdx.x;
	if (i > size) return;//think this is never true but oh well
	for (int j = threadIdx.x; j < size; j += blockDim.x) {
		GameObjectCUDA* object = &objs[i];
		GameObjectCUDA* other = &objs[j];
		//TRUE IS A COLLISION
		//if (!SphereCheck(object, other)) //simple sphere check to see if they are in range of each other
		//	continue;
		//full SAT to see if they actually collide
		//if collision found add to j.id to colliders
		if (SATCheck(object, other)) {
			unsigned int index = atomicInc(&objs[i].currentIdx, (MAX_COLLISIONS));//atomic inc prevents race conditions - each thread *should* always have its own unique index to access
			
			objs[i].colliders[index] = objs[j].id;
		}
	}
}
void MakePoints(JRrect rect,GameObjectCUDA& output) {
	for (int i = 0; i < 4; i++) {
		output.points[i] = make_float2(rect.points[i].x, rect.points[i].y);
	}
}
void MakeStructs(GameObjectCUDA* output, std::vector<GameObject*>& input) {
	for (int i = 0; i < input.size(); i++) {
		//clear colliders from last frame
		input[i]->colliders.clear();
		output[i].id = i;
		Vector2 pos = input[i]->GetPos();
		output[i].centre = make_float2(pos.x,pos.y);
		MakePoints(input[i]->GetCorners(), output[i]);
		input[i]->colliders.clear();
	}
}
void UnMakeStructs(std::vector<GameObject*>& output, GameObjectCUDA* input) {
	for (int i = 0; i < output.size(); i++) {
		int idx = static_cast<int>(input[i].currentIdx);
		for (int j = 0; j < std::min(idx, (int)output.size()); j++) {
			if (i == input[i].colliders[j])
				continue;
			if (output[i] == nullptr) 
				continue;
			int outID = input[i].colliders[j];
			if (outID < 0 || outID >= output.size()) {
				//GameLogging::GetInstance()->Log("ERROR: Invalid ID for detected collider (objectID colliderID) " + std::to_string(i) + " " + std::to_string(outID));
				continue;
			}

			output[i]->colliders.push_back(output[outID]);
		}
	}
}
void FreeCUDA() {
	if (allocated)
		hipFree(objsGPU);
	allocated = false;
}
void AllocCUDA(int size) {
	if (allocated)
		FreeCUDA();
	hipMalloc((void**)&objsGPU, sizeof(GameObjectCUDA) * size);
	allocated = true;
}
namespace JRCollision {
	void Init() {
		AllocCUDA(AllocSize);
	}
	void Free()
	{
		FreeCUDA();
	}
	void CalculateCollsion(std::vector<GameObject*>& UpdateQueue)
	{
		//make sure we have enough room on the GPU -- doing it this way prevents us from allocating every frame for different update queue sizes
		/*
		bool need_new_alloc = false;
		while (AllocSize < UpdateQueue.size()) {
			AllocSize += COLLISION_EXPAND_SIZE;
			need_new_alloc = true;
		}
		if (need_new_alloc || !allocated) {
			AllocCUDA(AllocSize);
		}
		*/
		AllocCUDA(UpdateQueue.size());
		GameObjectCUDA* objs = new GameObjectCUDA[UpdateQueue.size()];
		//copy to
		MakeStructs(objs, UpdateQueue);
		hipMemcpy(objsGPU, objs, sizeof(GameObjectCUDA) * UpdateQueue.size(), hipMemcpyHostToDevice);

		//exec
		GPUCollisionCalc << < UpdateQueue.size(), 1024 >> > (objsGPU, AllocSize);
		//copy from
		hipMemcpy(objs, objsGPU, sizeof(GameObjectCUDA) * UpdateQueue.size(), hipMemcpyDeviceToHost);
		UnMakeStructs(UpdateQueue, objs);
		//free
		delete[] objs;
		FreeCUDA();
	}
}