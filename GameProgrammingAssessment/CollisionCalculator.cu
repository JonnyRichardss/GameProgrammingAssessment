#include "hip/hip_runtime.h"
#include "CollisionCalculator.cuh"
#include "hip/hip_runtime.h"
#include ""
#include "Global_Flags.h"
#include "GameMath.h"
#include <math.h>
static int AllocSize = COLLISION_INIT_SIZE;

struct GameObjectCUDA {
	int id;
	float2 points[4];
	int colliders[MAX_COLLISIONS];
	unsigned int currentIdx = 0;
};
static GameObjectCUDA* objsGPU;
static bool allocated = false;
__global__ void GPUCollisionCalc(GameObjectCUDA* objs, int size) {
	int i = blockIdx.x;
	if (i > size) return;//think this is never true but oh well
	for (int j = threadIdx.x; j < size; j += blockDim.x) {
		//do SAT
		//if collision found add to j.id to colliders
		//currentIDx++
	}
}
void MakePoints(JRrect rect,GameObjectCUDA& output) {
	for (int i = 0; i < 4; i++) {
		output.points[i] = make_float2(rect.points[i].x, rect.points[i].y);
	}
}
void MakeStructs(GameObjectCUDA* output, std::vector<GameObject*> input) {
	for (int i = 0; i < input.size(); i++) {
		output[i].id = i;
		MakePoints(input[i]->GetCorners(), output[i]);
		input[i]->colliders.clear();
	}
}
void UnMakeStructs(std::vector<GameObject*> output, GameObjectCUDA* input) {
	for (int i = 0; i < output.size(); i++) {
		int idx = static_cast<int>(input[i].currentIdx);
		for (int j = 0; j < std::min(idx, (int)output.size()); j++) {
			if (i == input[i].colliders[j])continue;
			if (output[i] == nullptr) continue;
			output[i]->colliders.push_back(output[input[i].colliders[j]]);
		}
	}
}
void FreeCUDA() {
	if (allocated)
		hipFree(objsGPU);
	allocated = false;
}
void AllocCUDA(int size) {
	if (allocated)
		FreeCUDA();
	hipMalloc((void**)&objsGPU, sizeof(GameObjectCUDA) * size);
	allocated = true;
}

void CalculateCollsion(std::vector<GameObject*>& UpdateQueue)
{
	bool need_new_alloc = false;
	while (AllocSize > UpdateQueue.size()) {
		AllocSize += COLLISION_EXPAND_SIZE;
		need_new_alloc = true;
	}
	if (need_new_alloc) {
		AllocCUDA(AllocSize);
	}
	GameObjectCUDA* objs = new GameObjectCUDA[UpdateQueue.size()];
}